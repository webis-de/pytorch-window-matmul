#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/native/cuda/KernelUtils.cuh>

#include "utils.cuh"
#include <algorithm>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cmath>

#define _VOLATILE_
#define BLOCKSIZE 16

template <typename scalar_t>
__device__ void load(
    int thread_y, int thread_x, int b, int accessor_y, int accessor_x,
    torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> accessor,
    _VOLATILE_ scalar_t shared[BLOCKSIZE][BLOCKSIZE])
{
  if (accessor_y >= 0 && accessor_y < accessor.size(1) && accessor_x >= 0 && accessor_x < accessor.size(2))
    shared[thread_y][thread_x] = accessor[b][accessor_y][accessor_x];
  else
    shared[thread_y][thread_x] = 0;
}

template <typename scalar_t>
__device__ void compute_sub(
    _VOLATILE_ scalar_t x_shared[BLOCKSIZE][BLOCKSIZE],
    _VOLATILE_ scalar_t y_shared[BLOCKSIZE][BLOCKSIZE],
    scalar_t &sub)
{
  int thread_x = threadIdx.x;
  int thread_y = threadIdx.y;
#pragma unroll
  for (int block_idx = 0; block_idx < BLOCKSIZE; block_idx++)
    sub += x_shared[thread_x][block_idx] * y_shared[block_idx][thread_y];
}

template <typename scalar_t>
__global__ void window_matmul_kernel(
    torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> A_accessor,
    torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> B_accessor,
    torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> C_accessor,
    int window_size)
{
  // A: b x m x k
  // B: b x m x k
  // C: b x m x 2w + 1

  // Block index
  int block_x = blockIdx.x;
  int block_y = blockIdx.y;
  int batch_idx = blockIdx.z;

  // Thread index
  int thread_x = threadIdx.x;
  int thread_y = threadIdx.y;

  // Starting indices of A and B
  int a_m_start = BLOCKSIZE * block_y;
  int b_m_start = a_m_start + BLOCKSIZE * block_x - window_size;
  int a_m = a_m_start + thread_y;
  int b_m = b_m_start + thread_y;

  // ceil (K / BLOCKSIZE)
  int num_blocks = ceil(A_accessor.size(2) / (float)BLOCKSIZE);

  scalar_t c_sub = 0;
  for (int block_idx = 0; block_idx < num_blocks; block_idx++)
  {
    // Shared memory
    __shared__ scalar_t a_shared[BLOCKSIZE][BLOCKSIZE];
    __shared__ scalar_t b_shared[BLOCKSIZE][BLOCKSIZE];

    // Load the matrices into shared memory
    int block = block_idx * BLOCKSIZE;
    int ab_k = block + thread_x;
    load<scalar_t>(thread_y, thread_x, batch_idx, a_m, ab_k, A_accessor, a_shared);
    load<scalar_t>(thread_x, thread_y, batch_idx, b_m, ab_k, B_accessor, b_shared); // Transpose B
    __syncthreads();

    // Compute the partial product
    compute_sub<scalar_t>(a_shared, b_shared, c_sub);
    __syncthreads();
  }

  // Store the result in C
  int c_m = a_m_start + thread_x;
  int c_w = b_m_start + thread_y - c_m + window_size;
  if (c_m >= C_accessor.size(1) || c_w < 0 || c_w >= C_accessor.size(2))
    return;
  C_accessor[batch_idx][c_m][c_w] = c_sub;
}

template <typename scalar_t>
__global__ void unwindow_matmul_kernel_A(
    torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> A_accessor,
    torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> B_accessor,
    torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> C_accessor,
    int window_size)
{
  // A: b x m x 2w + 1
  // B: b x m x k
  // C: b x m x k

  // Block index
  int block_x = blockIdx.x;
  int block_y = blockIdx.y;
  int batch_idx = blockIdx.z;

  // Thread index
  int thread_x = threadIdx.x;
  int thread_y = threadIdx.y;

  // Starting indices of A and B
  int a_m_start = BLOCKSIZE * block_y;
  int b_k_start = BLOCKSIZE * block_x;
  int a_m = a_m_start + thread_y;
  int b_k = b_k_start + thread_x;

  int num_blocks;
  if (window_size < BLOCKSIZE && BLOCKSIZE <= A_accessor.size(1))
    // edge case when window_size < BLOCKSIZE <= m
    num_blocks = 2;
  else
    // ceil (2w+1 / BLOCKSIZE)
    num_blocks = ceil(A_accessor.size(2) / (float)BLOCKSIZE);

  scalar_t c_sub = 0;
  for (int block_idx = 0; block_idx < num_blocks; block_idx++)
  {
    // Shared memory
    __shared__ scalar_t a_shared[BLOCKSIZE][BLOCKSIZE];
    __shared__ scalar_t b_shared[BLOCKSIZE][BLOCKSIZE];

    // Load the matrices from global memory to shared memory
    int block = block_idx * BLOCKSIZE;
    int aw_idx = block + thread_x;
    int bw_idx = block + thread_y;
    int a_w = aw_idx - thread_y;
    int b_m = a_m_start + bw_idx - window_size;

    load<scalar_t>(thread_y, thread_x, batch_idx, a_m, a_w, A_accessor, a_shared);
    load<scalar_t>(thread_y, thread_x, batch_idx, b_m, b_k, B_accessor, b_shared);
    __syncthreads();

    // Compute the partial product
    compute_sub<scalar_t>(a_shared, b_shared, c_sub);
    __syncthreads();
  }

  // Store the result in C
  int c_m = a_m_start + thread_x;
  int c_k = b_k_start + thread_y;
  if (c_m >= C_accessor.size(1) || c_k >= C_accessor.size(2))
    return;
  C_accessor[batch_idx][c_m][c_k] = c_sub;
}

template <typename scalar_t>
__global__ void unwindow_matmul_kernel_B(
    torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> A_accessor,
    torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> B_accessor,
    torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> C_accessor,
    int window_size)
{
  // A: b x m x 2w + 1
  // B: b x m x k
  // C: b x m x k

  // Block index
  int block_x = blockIdx.x;
  int block_y = blockIdx.y;
  int batch_idx = blockIdx.z;

  // Thread index
  int thread_x = threadIdx.x;
  int thread_y = threadIdx.y;

  // Starting indices of A and B
  int a_m_start = BLOCKSIZE * block_y;
  int b_k_start = BLOCKSIZE * block_x;
  // int a_m = a_m_start + thread_y; // needs to use block_idx
  int b_k = b_k_start + thread_x;

  int num_blocks;
  if (window_size < BLOCKSIZE && BLOCKSIZE <= A_accessor.size(1))
    // edge case when window_size < BLOCKSIZE <= m
    num_blocks = 2;
  else
    // ceil (2w+1 / BLOCKSIZE)
    num_blocks = ceil(A_accessor.size(2) / (float)BLOCKSIZE);

  scalar_t c_sub = 0;
  for (int block_idx = 0; block_idx < num_blocks; block_idx++)
  {
    // Shared memory
    __shared__ scalar_t a_shared[BLOCKSIZE][BLOCKSIZE];
    __shared__ scalar_t b_shared[BLOCKSIZE][BLOCKSIZE];

    // Load the matrices from global memory to shared memory
    int block = block_idx * BLOCKSIZE;
    int a_m = a_m_start + thread_y - window_size + block;
    int aw_idx = block + thread_x;
    int bw_idx = block + thread_y;
    int a_w = aw_idx - thread_y + 2 * window_size - block * 2;
    int b_m = a_m_start + bw_idx - window_size;

    load<scalar_t>(thread_x, thread_y, batch_idx, a_m, a_w, A_accessor, a_shared); // transpose A
    load<scalar_t>(thread_y, thread_x, batch_idx, b_m, b_k, B_accessor, b_shared);
    __syncthreads();

    // Compute the partial product
    compute_sub<scalar_t>(a_shared, b_shared, c_sub);
    __syncthreads();
  }

  // Store the result in C
  int c_m = a_m_start + thread_x;
  int c_k = b_k_start + thread_y;
  if (c_m >= C_accessor.size(1) || c_k >= C_accessor.size(2))
    return;
  C_accessor[batch_idx][c_m][c_k] = c_sub;
}

dim3 get_grid(int b, int m, int window_size)
{
  dim3 grid;
  int num_w_blocks, num_m_blocks;
  if (window_size < BLOCKSIZE && BLOCKSIZE <= m)
    num_w_blocks = 2;
  else
    num_w_blocks = ceil((window_size * 2 + 1) / (float)BLOCKSIZE);
  num_m_blocks = ceil(m / (float)BLOCKSIZE);
  grid.x = num_w_blocks;
  grid.y = num_m_blocks;
  grid.z = b;
  return grid;
}

torch::Tensor window_matmul_fw_cuda(torch::Tensor A, torch::Tensor B, int window_size)
{
  CHECK_CUDA(A);
  CHECK_CUDA(B);

  CHECK_INPUT(A.dim() == B.dim());
  CHECK_INPUT(A.size(0) == B.size(0));
  CHECK_INPUT(A.size(1) == B.size(2));
  CHECK_INPUT(A.size(2) == B.size(1));

  // make contiguous
  A = A.contiguous();
  B = B.transpose(-1, -2).contiguous();

  // initialize output
  torch::Tensor C;
  auto sizes = A.sizes().vec();
  sizes[2] = window_size * 2 + 1;
  C = torch::zeros(sizes, A.options());

  // compute grid
  dim3 grid = get_grid(A.size(0), A.size(1), window_size);
  dim3 threads(BLOCKSIZE, BLOCKSIZE);

  // run kernel
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      C.scalar_type(), "window_matmul_fw_cuda", [&]
      {
        auto A_accessor = A.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>();
        auto B_accessor = B.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>();
        auto C_accessor = C.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>();
        window_matmul_kernel<scalar_t><<<grid, threads>>>(
          A_accessor,
          B_accessor,
          C_accessor,
          window_size
          ); });
  return C;
}

std::tuple<torch::Tensor, torch::Tensor> window_matmul_bw_cuda(
    torch::Tensor A, torch::Tensor B, int window_size, torch::Tensor grad_C)
{
  CHECK_CUDA(A);
  CHECK_CUDA(B);

  CHECK_INPUT(A.dim() == B.dim());
  CHECK_INPUT(A.size(0) == B.size(0));
  CHECK_INPUT(A.size(1) == B.size(2));
  CHECK_INPUT(A.size(2) == B.size(1));

  // make contiguous
  A = A.contiguous();
  B = B.transpose(-1, -2).contiguous();
  grad_C = grad_C.contiguous();

  // initialize output
  torch::Tensor grad_A, grad_B;
  grad_A = torch::zeros(A.sizes().vec(), A.options());
  grad_B = torch::zeros(B.sizes().vec(), B.options());

  // compute grid
  dim3 grid = get_grid(A.size(0), A.size(1), window_size);
  dim3 threads(BLOCKSIZE, BLOCKSIZE);

  // run kernel
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      grad_C.scalar_type(), "window_matmul_bw_cuda", [&]
      {
        auto A_accessor = A.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>();
        auto B_accessor = B.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>();
        auto grad_C_accessor = grad_C.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>();
        auto grad_A_accessor = grad_A.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>();
        auto grad_B_accessor = grad_B.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>();
        unwindow_matmul_kernel_A<scalar_t><<<grid, threads>>>(
          grad_C_accessor,
          B_accessor,
          grad_A_accessor,
          window_size);
        unwindow_matmul_kernel_B<scalar_t><<<grid, threads>>>(
          grad_C_accessor,
          A_accessor,
          grad_B_accessor,
          window_size); });
  return std::make_tuple(grad_A, grad_B.transpose(-1, -2));
}

torch::Tensor unwindow_matmul_fw_cuda(torch::Tensor A, torch::Tensor B, int window_size)
{
  CHECK_CUDA(A);
  CHECK_CUDA(B);

  CHECK_INPUT(A.dim() == B.dim());
  CHECK_INPUT(A.size(0) == B.size(0));
  CHECK_INPUT(A.size(1) == B.size(1));
  CHECK_INPUT(A.size(2) == window_size * 2 + 1);

  // make contiguous
  A = A.contiguous();
  B = B.contiguous();

  // initialize output
  torch::Tensor C;
  auto sizes = B.sizes().vec();
  C = torch::zeros(sizes, A.options());

  // compute grid
  dim3 grid = get_grid(A.size(0), A.size(1), window_size);
  dim3 threads(BLOCKSIZE, BLOCKSIZE);

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      C.scalar_type(), "unwindow_matmul_fw_cuda", [&]
      {
        auto A_accessor = A.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>();
        auto B_accessor = B.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>();
        auto C_accessor = C.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>();

        unwindow_matmul_kernel_A<scalar_t><<<grid, threads>>>(
          A_accessor,
          B_accessor,
          C_accessor,
          window_size); });
  return C;
}

std::tuple<torch::Tensor, torch::Tensor> unwindow_matmul_bw_cuda(
    torch::Tensor A, torch::Tensor B, int window_size, torch::Tensor grad_output)
{
  CHECK_CUDA(A);
  CHECK_CUDA(B);

  CHECK_INPUT(A.dim() == B.dim());
  CHECK_INPUT(A.size(0) == B.size(0));
  CHECK_INPUT(A.size(1) == B.size(1));
  CHECK_INPUT(A.size(2) == window_size * 2 + 1);

  A = A.contiguous();
  B = B.contiguous();
  grad_output = grad_output.contiguous();

  // initialize output
  torch::Tensor grad_A, grad_B;
  grad_A = torch::zeros(A.sizes().vec(), A.options());
  grad_B = torch::zeros(B.sizes().vec(), B.options());

  // compute grid
  dim3 grid = get_grid(A.size(0), A.size(1), window_size);
  dim3 threads(BLOCKSIZE, BLOCKSIZE);

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      grad_A.scalar_type(), "unwindow_matmul_bw_cuda", [&]
      {
        auto A_accessor = A.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>();
        auto B_accessor = B.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>();
        auto grad_C_accessor = grad_output.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>();
        auto grad_A_accessor = grad_A.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>();
        auto grad_B_accessor = grad_B.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>();

        window_matmul_kernel<scalar_t><<<grid, threads>>>(
          grad_C_accessor,
          B_accessor,
          grad_A_accessor,
          window_size
          );
        unwindow_matmul_kernel_B<scalar_t><<<grid, threads>>>(
          A_accessor,
          grad_C_accessor,
          grad_B_accessor,
          window_size
          ); });
  return std::make_tuple(grad_A, grad_B);
}
